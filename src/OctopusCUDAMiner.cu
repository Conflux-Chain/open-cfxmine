#include "hip/hip_runtime.h"
#include "OctopusCUDAMiner.h"
#include "StratumClient.h"
#include "cuda/octopus.cuh"
#include "cuda/precomputation.h"
#include "hex.h"
#include "light.h"
#include "octopus_params.h"
#include "octopus_structs.h"

#include <functional>
#include <iostream>

#define checkCudaErrors(call)                                                  \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (hipSuccess != err) {                                                  \
      fprintf(stderr, "CUDA error RUNTIME: '%d' in func '%s' line %d\n", err,  \
              __FUNCTION__, __LINE__);                                         \
      abort();                                                                 \
    }                                                                          \
  } while (0)

class CUDADagManager {
public:
  void reset(uint64_t blockHeight) {
    dagSize = octopus_get_datasize(blockHeight);
    dagNumItems = dagSize / OCTOPUS_MIX_BYTES;
    lightSize = octopus_get_cachesize(blockHeight);
    lightNumItems = lightSize / OCTOPUS_HASH_BYTES;
    if (memoryDagSize < dagSize) {
      if (h_dag) {
        checkCudaErrors(hipFree(h_dag));
      }
      {
        hipError_t err = hipMalloc(&h_dag, dagSize);
        if (hipSuccess != err) {
          if (hipErrorOutOfMemory == err) {
            fprintf(stderr, "hipMalloc failed. Reason: Insufficient memory\n");
          } else {
            fprintf(stderr, "CUDA error RUNTIME: '%d' in func '%s' line %d",
                    err, __FUNCTION__, __LINE__);
          }
          abort();
        }
      }
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_dag), &h_dag, sizeof(void *)));
      memoryDagSize = dagSize;
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), &dagNumItems, sizeof(u32)));
    if (memoryLightSize < lightSize) {
      if (h_light) {
        checkCudaErrors(hipFree(h_light));
      }
      checkCudaErrors(hipMalloc(&h_light, lightSize));
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_light), &h_light, sizeof(void *)));
      memoryLightSize = lightSize;
    }
    checkCudaErrors(
        hipMemcpyToSymbol(HIP_SYMBOL(d_light_size), &lightNumItems, sizeof(u32)));
    checkCudaErrors(hipDeviceSynchronize());
  }

  void FreeCUDA() {
    if (h_dag) {
      checkCudaErrors(hipFree(h_dag));
    }
    if (h_light) {
      checkCudaErrors(hipFree(h_light));
    }
  }

public:
  void *h_light = 0;

  u32 lightNumItems;
  size_t lightSize;
  u32 dagNumItems;
  size_t dagSize;

private:
  void *h_dag = 0;

  size_t memoryDagSize = 0;
  size_t memoryLightSize = 0;
};

OctopusCUDAMiner::ThreadContext::ThreadContext(OctopusCUDAMiner *miner_,
                                               int device_id_, int context_id_)
    : miner(miner_), device_id(device_id_), context_id(context_id_),
      dagManager(new CUDADagManager()) {}

OctopusCUDAMiner::OctopusCUDAMiner(const OctopusCUDAMinerSettings &settings)
    : AbstractMiner(), settings(settings) {
  int device_count;
  checkCudaErrors(hipGetDeviceCount(&device_count));

  int context_id = 0;

  for (int device_id : settings.device_ids) {
    if (device_id < device_count) {
      device_ids.push_back(device_id);
      threadContexts.emplace_back(this, device_id, context_id++);
    } else {
      std::cerr << "CUDA device_id = " << device_id << " does not exist."
                << std::endl;
    }
  }

  if (device_ids.empty()) {
    abort();
  }
}

OctopusCUDAMiner::~OctopusCUDAMiner() {}

void OctopusCUDAMiner::Start() {
  workerThreads = std::make_unique<boost::thread_group>();
  for (size_t i = 0; i < threadContexts.size(); ++i) {
    workerThreads->create_thread(
        boost::bind(&OctopusCUDAMiner::Work, this, &threadContexts[i]));
  }
}

void OctopusCUDAMiner::ThreadContext::InitCUDA() {
  checkCudaErrors(hipSetDevice(device_id));
  checkCudaErrors(hipHostMalloc(&d_search_results, sizeof(SearchResults)));
}

void OctopusCUDAMiner::ThreadContext::InitPerEpoch(uint64_t blockHeight) {
  dagManager->reset(blockHeight);
  auto h_light = octopus_light_new(blockHeight);
  checkCudaErrors(hipMemcpy(dagManager->h_light, h_light->cache,
                             dagManager->lightSize, hipMemcpyHostToDevice));
  octopus_light_delete(h_light);

  const uint32_t work = dagManager->dagSize / 8;
  const uint32_t run = miner->settings.initGridSize * INIT_BLOCK_SIZE;

  uint32_t base;
  for (base = 0; base <= work - run; base += run) {
    InitDagItems<<<miner->settings.initGridSize, INIT_BLOCK_SIZE>>>(base);
  }
  if (base < work) {
    const uint32_t lastGrid =
        ((work - base) + INIT_BLOCK_SIZE - 1) / INIT_BLOCK_SIZE;
    InitDagItems<<<lastGrid, INIT_BLOCK_SIZE>>>(base);
  }
  checkCudaErrors(hipDeviceSynchronize());
}

void OctopusCUDAMiner::ThreadContext::InitPerHeader(
    const octopus_h256_t headerHash, const octopus_h256_t boundary) {
  checkCudaErrors(
      hipMemcpyToSymbol(HIP_SYMBOL(d_header), headerHash.b, sizeof(headerHash)));
  {
    uint64_t buffer[4];
    for (int i = 0; i < 4; ++i) {
      const uint64_t b = reinterpret_cast<const uint64_t *>(boundary.b)[i];
      buffer[i] = ((b & 0xff00000000000000ULL) >> 56) |
                  ((b & 0x00ff000000000000ULL) >> 40) |
                  ((b & 0x0000ff0000000000ULL) >> 24) |
                  ((b & 0x000000ff00000000ULL) >> 8) |
                  ((b & 0x00000000ff000000ULL) << 8) |
                  ((b & 0x0000000000ff0000ULL) << 24) |
                  ((b & 0x000000000000ff00ULL) << 40) |
                  ((b & 0x00000000000000ffULL) << 56);
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_boundary), buffer, sizeof(boundary)));
  }
  OctopusABCW p(headerHash);
  const u32 a = p.a;
  const u32 b = p.b;
  const u32 c = p.c;
  const u32 w = p.w;
  Precomputation<OCTOPUS_N> pre(a, b, c, w);
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_x), pre.x, sizeof(u32) * OCTOPUS_N));
  checkCudaErrors(hipDeviceSynchronize());
}

void OctopusCUDAMiner::Work(ThreadContext *ctx) {
  ctx->InitCUDA();

  const uint32_t searchGridSize = settings.searchGridSize;
  const uint32_t batchSize = searchGridSize * SEARCH_BLOCK_SIZE;

  std::string jobId;
  uint64_t blockHeight = std::numeric_limits<uint64_t>::max();
  std::string headerHashString;
  octopus_h256_t headerHash;
  octopus_h256_t boundary;
  uint64_t nonce = ctx->context_id * batchSize;

  while (is_running.load(std::memory_order_acquire)) {
    if (workJobId == MINER_NO_WORK) {
      boost::this_thread::sleep_for(boost::chrono::milliseconds(5000));
      continue;
    }
    if (0 != memcmp(headerHash.b, workHeaderHash.b, sizeof(headerHash))) {
      jobId = workJobId;
      headerHashString = workHeaderHashString;
      if (octopus_get_epoch(blockHeight) !=
          octopus_get_epoch(workBlockHeight)) {
        ctx->InitPerEpoch(workBlockHeight);
        blockHeight = workBlockHeight;
      }
      ctx->InitPerHeader(workHeaderHash, workBoundary);
      memcpy(headerHash.b, workHeaderHash.b, sizeof(headerHash));
      memcpy(boundary.b, workBoundary.b, sizeof(boundary));
      nonce = ctx->context_id * batchSize;
    }

    volatile SearchResults &search_results =
        *reinterpret_cast<SearchResults *>(ctx->d_search_results);
    search_results.count = 0;
    Compute<<<settings.searchGridSize, SEARCH_BLOCK_SIZE>>>(
        nonce, reinterpret_cast<SearchResults *>(ctx->d_search_results));
    checkCudaErrors(hipDeviceSynchronize());

    uint32_t found_count =
        std::min((uint32_t)search_results.count, MAX_SEARCH_RESULTS);
    for (uint32_t i = 0; i < found_count; i++) {
      uint64_t found_nonce = nonce + search_results.result[i].nonce_offset;
      std::vector<std::string> solutions;
      solutions.push_back(jobId);
      solutions.push_back("0x" + hex::to_hex_string(found_nonce));
      solutions.push_back(headerHashString);
      client->OnSolutionFound(solutions);
    }
    client->UpdateHashRate(batchSize);
    nonce += batchSize * device_ids.size();
  }

  checkCudaErrors(hipDeviceSynchronize());
  ctx->dagManager->FreeCUDA();
  checkCudaErrors(hipHostFree(ctx->d_search_results));
}
